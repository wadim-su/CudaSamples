#include "DeviceCalculations.cuh"

#include <thrust/device_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>

#include <hiprand.h>


struct inside_circle
{
  __device__ int8_t operator()(const thrust::tuple<float, float>& p) const
  {
    return (((thrust::get<0>(p) - 0.5) * (thrust::get<0>(p) - 0.5) + (thrust::get<1>(p) - 0.5) * (thrust::get<1>(p) - 0.5)) < 0.25) ? 1 : 0;
  }
};

__host__
size_t calc_on_device(size_t numberOfPoints)
{
  thrust::device_vector<float> pointsX(numberOfPoints);
  thrust::device_vector<float> pointsY(numberOfPoints);

  // Generate random points using cuRAND
  hiprandGenerator_t generator;
  hiprandCreateGenerator(&generator, /*HIPRAND_RNG_QUASI_DEFAULT*/HIPRAND_RNG_PSEUDO_DEFAULT);

  hiprandGenerateUniform(generator, thrust::raw_pointer_cast(pointsX.data()), numberOfPoints);
  hiprandGenerateUniform(generator, thrust::raw_pointer_cast(pointsY.data()), numberOfPoints);

  // Count points inside circle using reduction from Thrust
  thrust::device_vector<int8_t> insideCircle(numberOfPoints);

  auto first = thrust::make_zip_iterator(thrust::make_tuple(pointsX.begin(), pointsY.begin()));
  auto last  = thrust::make_zip_iterator(thrust::make_tuple(pointsX.end()  , pointsY.end()  ));

  thrust::transform(first, last, insideCircle.begin(), inside_circle());
  size_t total = thrust::reduce(insideCircle.begin(), insideCircle.end(), (size_t)0, thrust::plus<size_t>());

  return total;
}
